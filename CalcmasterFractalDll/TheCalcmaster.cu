#include "hip/hip_runtime.h"
// Copyright 2024 - Gregory James Carriveau a.k.a. fractalsnakes840
// MIT License
// 
// Permission is hereby granted, free of charge, to any person obtaining a copy of this
// software and associated documentation files(the �Software�), to deal in the Software
// without restriction, including without limitation the rights to use, copy, modify,
// merge, publish, distribute, sublicense, and /or sell copies of the Software, and to
// permit persons to whom the Software is furnished to do so, subject to the following conditions :
//
// The above copyright notice and this permission notice shall be included in all copies
// or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED �AS IS�, WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED,
// INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR
// PURPOSE AND NONINFRINGEMENT.IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE
// FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR
// OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
// DEALINGS IN THE SOFTWARE.

// Let's tickle the CUDAs :)
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>	// not required.. I use it for getting rid of Intellisense squigglies under blockIdx, blockDim, threadIdx in Visual Studio 2022
#include <thrust/complex.h>             // numerics for double precision complex numbers

// Global device constants
__device__ __constant__ double g_e{ 2.718281828459045 }; // euler's number

// Global device variables
__device__ const int g_colorsInPalette{ 1000 };
__device__ double g_juliaCenterX;
__device__ double g_juliaCenterY;
__device__ int    g_maxIts;
__device__ double g_limit;
__device__ int    g_N;
__device__ int    g_ismove;
typedef thrust::complex<double> (*frmptr)(thrust::complex<double>, thrust::complex<double>);
__device__ frmptr g_alg;

// Fractal default 0: z^2 + p
__device__ thrust::complex<double> frmMandelbrot(thrust::complex<double> z, thrust::complex<double> p)
{
    return z * z + p;
}
// Fractal 1: (sin(z) * z)^2  + p
__device__ thrust::complex<double> frmSinPow2(thrust::complex<double> z, thrust::complex<double> p)
{
    thrust::complex<double>temp{ thrust::sin(z) * z };
    return temp * temp + p;
}
// Fractal 2: (sin(z) * z)^3  + p
__device__ thrust::complex<double> frmSinPow3(thrust::complex<double> z, thrust::complex<double> p)
{
    thrust::complex<double>temp{ thrust::sin(z) * z };
    return temp * temp * temp + p;
}
// Fractal 3: (sin(z) * z)^4  + p
__device__ thrust::complex<double> frmSinPow4(thrust::complex<double> z, thrust::complex<double> p)
{
    thrust::complex<double>temp{ thrust::sin(z) * z };
    return temp * temp * temp * temp + p;
}
// Fractal 4: (sin(z) * z / div)^2 + p
__device__ thrust::complex<double> frmSinPow2Div1(thrust::complex<double> z, thrust::complex<double> p)
{
    thrust::complex<double>temp{ z / thrust::complex<double>{4.0, -1.5} };
    temp *= thrust::sin(z);
    temp *= temp;
    return temp + p;
}
// Fractal 5: (cos(z) * z)^2  + p
__device__ thrust::complex<double> frmCosPow2(thrust::complex<double> z, thrust::complex<double> p)
{
    thrust::complex<double>temp{ thrust::cos(z) * z };
    return temp * temp + p;
}
// Fractal 6: ((z^2 + p) * (cos(z) * z)^2 + p)
__device__ thrust::complex<double> frmCosPow2MandelbrotHybrid(thrust::complex<double> z, thrust::complex<double> p)
{
    thrust::complex<double>temp{ z * z + p};
    thrust::complex<double>temp2{ thrust::cos(z) * z };
    temp2 = temp2 * temp2 + p;
    return temp * temp2;
}
// Fractal 7: Mandelbrot => Burning Ship (+re, -im) => (cos(Burning Ship) * (Burning Ship))^2 + p
__device__ thrust::complex<double> frmCosPow2AbsRIMandelbrotHybrid(thrust::complex<double> z, thrust::complex<double> p)
{
    thrust::complex<double>temp{ thrust::cos(z) * z };
    // this is the conjugate of Mandelbrot z
    temp = thrust::complex<double>{ temp.real(), -temp.imag() };
    // Hybridization of Burning Ship
    temp = thrust::cos(temp) * temp; // cos(+re,-im) * (+re,-im)
    return temp * temp + p;
}
// Fractal 8: (cos(+re,-im) * (+re,-im))^2 + p
__device__ thrust::complex<double> frmCosPow2AbsRI(thrust::complex<double> z, thrust::complex<double> p)
{
    thrust::complex<double> temp{ abs(z.real()), -abs(z.imag()) };
    // this is the conjugate of Mandelbrot z
    temp = thrust::cos(temp) * temp;
    return temp * temp + p;
}
// Fractal 28: SinAbsRI
__device__ thrust::complex<double> frmSinAbsRI(thrust::complex<double> z, thrust::complex<double> p)
{
    thrust::complex<double> temp{ abs(z.real()), -abs(z.imag()) };
    temp = thrust::sin(temp) * temp * temp;
    return temp + p;
}
// Fractal 9: Experiment number 1 ((1 - z^3) / 6) / ((1 - z - z^2) / 2)^2 + p
__device__ thrust::complex<double> frmExperiment1(thrust::complex<double> z, thrust::complex<double> p)
{
    thrust::complex<double> temp{ (1 - z * z * z) / 6 };
    thrust::complex<double> temp2{ (1 - z - z * z) / 2 };
    temp /= temp2 * temp2;
    return temp + p;
}
// Fractal 10: CosPow2AbsRIPow4
__device__ thrust::complex<double> frmCosPow2AbsRIPow4(thrust::complex<double> z, thrust::complex<double> p)
{
    // res = Complex.Pow(new Complex(Math.Abs(res.Real), -Math.Abs(res.Imaginary)), 4);
    // return Complex.Pow(Complex.Multiply(Complex.Cos(res), res), 2.0) + p;
    thrust::complex<double> temp{ thrust::pow(thrust::complex<double>{cuda::std::abs(z.real()), -cuda::std::abs(z.imag())}, 4)};
    temp = thrust::pow(thrust::cos(temp) * temp, 2);
    return temp + p;
}
// Fractal 11: CosPow2SinPow2Hybrid
__device__ thrust::complex<double> frmCosPow2SinPow2Hybrid(thrust::complex<double> z, thrust::complex<double> p)
{
    // res = Complex.Pow(Complex.Multiply(Complex.Sin(res), res), 2.0) + p;
    // return Complex.Pow(Complex.Multiply(Complex.Cos(res), res), 2.0) + p;
    thrust::complex<double> temp{ (thrust::sin(z) * z) };
    temp *= temp;
    temp += p;
    temp = thrust::cos(temp) * temp;
    temp *= temp;
    return temp + p;
}
// Fractal 12: CosPow3
__device__ thrust::complex<double> frmCosPow3(thrust::complex<double> z, thrust::complex<double> p)
{
    // return return Complex.Pow(Complex.Multiply(Complex.Cos(res), res), 3.0) + p;
    thrust::complex<double> temp{ (thrust::cos(z) * z) };
    temp *= temp * temp;
    return temp + p;
}
// Fractal 13: WeirdLim5
__device__ thrust::complex<double> frmWeirdLim5(thrust::complex<double> z, thrust::complex<double> p)
{
    // return Complex.Pow(Complex.Multiply(Complex.Cos(res), Complex.Divide(res, Complex.Add(res, -1))), 3.0) + p;
    thrust::complex<double> temp{ thrust::cos(z) * z / (z - 1.0) };
    temp = temp * temp * temp;
    return temp + p;
}
// Fractal 14: WeirdLim9 (same as above, but with a higher escape threshold)
__device__ thrust::complex<double> frmWeirdLim9(thrust::complex<double> z, thrust::complex<double> p)
{
    // return Complex.Pow(Complex.Multiply(Complex.Cos(res), Complex.Divide(res, Complex.Add(res, -1))), 3.0) + p;
    thrust::complex<double> temp{ thrust::cos(z) * z / (z - 1.0) };
    temp = temp * temp * temp;
    return temp + p;
}
// Fractal 15: Weird2
__device__ thrust::complex<double> frmWeird2(thrust::complex<double> z, thrust::complex<double> p)
{
    // return Complex.Add(Complex.Add(Complex.Sin(res), Complex.Pow(Math.E, res)), p);
    thrust::complex<double> temp{ thrust::sin(z) };
    temp += thrust::pow(g_e, z);
    return temp + p;
}
// Fractal 16: Weird3
__device__ thrust::complex<double> frmWeird3(thrust::complex<double> z, thrust::complex<double> p)
{
    // return Complex.Pow(Complex.Multiply(Complex.Pow(Complex.Cos(Z), 3.0), Z), 2.0) + p;
    thrust::complex<double> temp{ thrust::cos(z) };
    temp *= temp;
    temp *= temp;
    temp *= z;
    return temp + p;
}
// Fractal 17: Weird4
__device__ thrust::complex<double> frmWeird4(thrust::complex<double> z, thrust::complex<double> p)
{
    // return Complex.Pow(Complex.Multiply(Complex.Pow(Complex.Cos(res), 4.0), res), 2.0) + p;
    thrust::complex<double> temp{ thrust::pow(thrust::cos(z),4) };
    temp *= z;
    temp *= temp;
    return temp + p;
}
// Fractal 18: Weird5
__device__ thrust::complex<double> frmWeird5(thrust::complex<double> z, thrust::complex<double> p)
{
    // return Complex.Pow(Complex.Multiply(Complex.Pow(Complex.Sin(res), 4.0), res), 2.0) + p;
    thrust::complex<double> temp{ thrust::pow(thrust::sin(z),4) * z };
    temp *= temp;
    return temp + p;
}
// Fractal 19: Weird6
__device__ thrust::complex<double> frmWeird6(thrust::complex<double> z, thrust::complex<double> p)
{
    // This one is really cool.  This is art.
    // Complex cosReal = new Complex(Math.Cos(res.Real), res.Imaginary);
    // return Complex.Pow(Complex.Multiply(Complex.Pow(cosReal, 4.0), res), 2.0) + p;
    thrust::complex<double> cosREAL{ cos(z.real()), z.imag() };
    thrust::complex<double> temp{ thrust::pow(thrust::pow(cosREAL, 4) * z, 2) };
    return temp + p;
}
// Fractal 20: Weird7
__device__ thrust::complex<double> frmWeird7(thrust::complex<double> z, thrust::complex<double> p)
{
    // This one is really cool.  This is art.
    // Complex cosReal = new Complex(Math.Cos(res.Real), Math.Sin(res.Imaginary));
    // return Complex.Pow(Complex.Multiply(Complex.Pow(Complex.Cos(res), 4.0), cosReal), 2.0) + p;
    thrust::complex<double> cosREALsinIMAG{ cos(z.real()), sin(z.imag()) };
    thrust::complex<double> temp{thrust::cos(z)};
    temp = thrust::pow(thrust::pow(temp,4) * cosREALsinIMAG,2);
    return temp + p;
}
// Fractal 21: Weird8
__device__ thrust::complex<double> frmWeird8(thrust::complex<double> z, thrust::complex<double> p)
{
    // return Complex.Add(Complex.Pow(Complex.Divide(res,Complex.Pow(new Complex(1, -0.5),res)), 2.0), p);
    thrust::complex<double> temp{ 1, -0.5 };
    temp = thrust::pow(temp, z);
    temp = thrust::pow(z / temp, 2);
    return temp + p;
}
// Fractal 22: Weird9
__device__ thrust::complex<double> frmWeird9(thrust::complex<double> z, thrust::complex<double> p)
{
    /*
    double oneThird = 1.0 / 3.0;
    return Complex.Subtract(
        Complex.Add(
            Complex.Subtract(
                Complex.Multiply(0.5,Complex.Pow(res,2)),
                Complex.Multiply(oneThird, Complex.Pow(res, 3))
            ),
            res
        ),
    p);
    */
    const double oneThird{ 1.0 / 3.0 };
    thrust::complex<double> temp{ z * z * 0.5 };
    temp -= z * z * z * oneThird;
    temp += z;
    return temp - p;
}
// Fractal 23: Weird10
__device__ thrust::complex<double> frmWeird10(thrust::complex<double> z, thrust::complex<double> p)
{
    /*
    double oneThird = 1.0 / 3.0;
    return Complex.Subtract(
        Complex.Add(
            Complex.Subtract(
                Complex.Multiply(0.5, Complex.Pow(res, 2)),
                Complex.Multiply(oneThird, Complex.Pow(Complex.Cos(res), 3))
            ),
            res
        ),
    p);
    */
    const double oneThird{ 1.0 / 3.0 };
    thrust::complex<double> temp{ z * z * 0.5 };
    temp -= thrust::pow(thrust::cos(z),3) * oneThird;
    temp += z;
    return temp - p;
}
// Fractal 24: Weird11
__device__ thrust::complex<double> frmWeird11(thrust::complex<double> z, thrust::complex<double> p)
{
    /*
    double oneThird = 1.0 / 3.0;
    return Complex.Add(
        Complex.Pow((1 / Complex.Tan(res)),Complex.Multiply(oneThird,res)),
    p);
    */
    const double oneThird{ 1.0 / 3.0 };
    thrust::complex<double> temp{ z * oneThird };
    temp = thrust::pow((1 / thrust::tan(z)), temp);
    return temp + p;
}
// Fractal 25: Mandelbrot4th
__device__ thrust::complex<double> frmMandelbrot4th(thrust::complex<double> z, thrust::complex<double> p)
{
    // return Complex.Pow(Complex.Add(Complex.Pow(res, 2.0), p),4.0);
    thrust::complex<double> temp{ z * z };
    temp += p;
    return thrust::pow(temp, 4);
}
// Fractal 26: Mandelbrot8th
__device__ thrust::complex<double> frmMandelbrot8th(thrust::complex<double> z, thrust::complex<double> p)
{
    // return Complex.Pow(Complex.Add(Complex.Pow(res, 2.0), p),8.0);
    thrust::complex<double> temp{ z * z };
    temp += p;
    return thrust::pow(temp,8);
}
// Fractal 27: BurningShip
__device__ thrust::complex<double> frmBurningShip(thrust::complex<double> z, thrust::complex<double> p)
{
    // return Complex.Add(Complex.Pow(new Complex(Math.Abs(res.Real),-Math.Abs(res.Imaginary)) , 2.0), p);
    thrust::complex<double> temp{ cuda::std::abs(z.real()), -cuda::std::abs(z.imag())};
    temp *= temp;
    //temp *= z;
    return temp + p;
}
// Fracta 29: Experiment2
__device__ thrust::complex<double> frmExperiment2(thrust::complex<double> z, thrust::complex<double> p)
{
    // return Complex.Add(Complex.Pow(new Complex(Math.Abs(res.Real),-Math.Abs(res.Imaginary)) , 2.0), p);
    thrust::complex<double> temp{ cuda::std::abs(z.real()), z.imag()};
    temp = temp + thrust::pow(z,2);
    //temp *= z;
    return temp + p;
}
// Fracta 30: Experiment3
__device__ thrust::complex<double> frmExperiment3(thrust::complex<double> z, thrust::complex<double> p)
{
    // return Complex.Add(Complex.Pow(new Complex(Math.Abs(res.Real),-Math.Abs(res.Imaginary)) , 2.0), p);
    thrust::complex<double> temp{ cuda::std::abs(z.real()), z.imag() };
    temp = temp + thrust::pow(z, 2) + thrust::pow(z, 3) / 1.8;
    //temp *= z;
    return temp + p;
}
// Fracta 31: Experiment4
__device__ thrust::complex<double> frmExperiment4(thrust::complex<double> z, thrust::complex<double> p)
{
    // return Complex.Add(Complex.Pow(new Complex(Math.Abs(res.Real),-Math.Abs(res.Imaginary)) , 2.0), p);
    thrust::complex<double> temp{ thrust::pow(z,1.333333333) };
    //temp *= z;
    return temp + thrust::cos(p);
}
// Fracta 32: Experiment5
__device__ thrust::complex<double> frmExperiment5(thrust::complex<double> z, thrust::complex<double> p)
{
    // return Complex.Add(Complex.Pow(new Complex(Math.Abs(res.Real),-Math.Abs(res.Imaginary)) , 2.0), p);
    thrust::complex<double> temp{ thrust::pow(z,1.666666666) };
    //temp *= z;
    return temp + thrust::sin(z + p);
}
// Fractal 33: Experiment 6
__device__ thrust::complex<double> frmExperiment6(thrust::complex<double> z, thrust::complex<double> p)
{
    return thrust::pow(z,3) - z + p;
}



__global__ void setTheDeviceGlobals(double juliaCenterX, double juliaCenterY, int maxIts, double limit, int fractalFormulaID, int N, int ismove)
{
    g_juliaCenterX = juliaCenterX;
    g_juliaCenterY = juliaCenterY;
    g_maxIts = maxIts;
    g_limit = limit;
    g_N = N;
    g_ismove = ismove;
    switch (fractalFormulaID)
    {
    case 1:
        g_alg = frmSinPow2;
        break;
    case 2:
        g_alg = frmSinPow3;
        break;
    case 3:
        g_alg = frmSinPow4;
        break;
    case 4:
        g_alg = frmSinPow2Div1;
        break;
    case 5:
        g_alg = frmCosPow2;
        break;
    case 6:
        g_alg = frmCosPow2MandelbrotHybrid;
        break;
    case 7:
        g_alg = frmCosPow2AbsRIMandelbrotHybrid;
        break;
    case 8:
        g_alg = frmCosPow2AbsRI;
        break;
    case 9:
        g_alg = frmExperiment1;
        break;
    case 10:
        g_alg = frmCosPow2AbsRIPow4;
        break;
    case 11:
        g_alg = frmCosPow2SinPow2Hybrid;
        break;
    case 12:
        g_alg = frmCosPow3;
        break;
    case 13:
        g_alg = frmWeirdLim5;
        break;
    case 14:
        g_alg = frmWeirdLim9;
        break;
    case 15:
        g_alg = frmWeird2;
        break;
    case 16:
        g_alg = frmWeird3;
        break;

    case 17:
        g_alg = frmWeird4;
        break;
    case 18:
        g_alg = frmWeird5;
        break;
    case 19:
        g_alg = frmWeird6;
        break;
    case 20:
        g_alg = frmWeird7;
        break;
    case 21:
        g_alg = frmWeird8;
        break;
    case 22:
        g_alg = frmWeird9;
        break;
    case 23:
        g_alg = frmWeird10;
        break;
    case 24:
        g_alg = frmWeird11;
        break;
    case 25:
        g_alg = frmMandelbrot4th;
        break;
    case 26:
        g_alg = frmMandelbrot8th;
        break;
    case 27:
        g_alg = frmBurningShip;
        break;
    case 28:
        g_alg = frmSinAbsRI;
        break;
    case 29:
        g_alg = frmExperiment2;
        break;
    case 30:
        g_alg = frmExperiment3;
        break;
    case 31:
        g_alg = frmExperiment4;
        break;
    case 32:
        g_alg = frmExperiment5;
        break;
    case 33:
        g_alg = frmExperiment6;
        break;
    default:
        g_alg = frmMandelbrot;
        break;
    }
}

__global__ void algMap(const double* __restrict__ realCoords, const double* __restrict__ imagCoords, int* __restrict__ iterations)
{
    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Boundary check
    if (tid < g_N)
    {
        // if we're doing a move UP DOWN LEFT or RIGHT, skip the non-zero iterations
        if (g_ismove == 1 && iterations[tid] != 0) return;
        thrust::complex<double> z{ 0.0,0.0 };
        const thrust::complex<double> p{ realCoords[tid], imagCoords[tid] };
        int i{ 0 };
        while (i < g_maxIts)
        {
            thrust::complex<double> temp{ g_alg(z, p)};
            if (thrust::abs(temp) > g_limit * g_limit) break;   // magnitude escapes the limit?
            z = temp;
            ++i;
        }
        iterations[tid] = i;
    }
}

__global__ void algJulia(const double* __restrict__ realCoords, const double* __restrict__ imagCoords, int* __restrict__ iterations)
{
    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Boundary check
    if (tid < g_N)
    {
        // if we're doing a move UP DOWN LEFT or RIGHT, skip the non-zero iterations
        if (g_ismove == 1 && iterations[tid] != 0) return;
        thrust::complex<double> z{ realCoords[tid], imagCoords[tid] };
        const thrust::complex<double> p{ g_juliaCenterX, g_juliaCenterY };
        int i{ 0 };
        while (i < g_maxIts)
        {
            thrust::complex<double> temp{ g_alg(z, p) };
            if (thrust::abs(temp) > g_limit * g_limit) break;   // magnitude escapes the limit?
            z = temp;
            ++i;
        }
        iterations[tid] = i;
    }
}

// Pretty leaves and scary hairy eyeball videos in areas near black holes
__global__ void algTheCalcmasterTwist(const double* __restrict__ realCoords, const double* __restrict__ imagCoords, int* __restrict__ iterations)
{
    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Boundary check
    if (tid < g_N)
    {
        // if we're doing a move UP DOWN LEFT or RIGHT, skip the non-zero iterations
        if (g_ismove == 1 && iterations[tid] != 0) return;
        thrust::complex<double> z{ realCoords[tid], imagCoords[tid] };
        const thrust::complex<double> p{ g_juliaCenterX, g_juliaCenterY };
        thrust::complex<double> temp{ 0.0, 0.0 };
        int i{ 0 };
        while (i < g_maxIts)
        {
            temp = { g_alg(z, p) };
            if (thrust::abs(temp) > g_limit * g_limit) break;   // magnitude escapes the limit?
            z = temp;
            ++i;
        }
        // iterations[tid] = i;
        
        // C#
        // return (Int32)
        // (
        //   Complex.Tanh(
        //     Complex.Subtract(
        //       Math.Min(res.Real, res.Imaginary), p
        //     )
        //   )
        //   .Magnitude * its
        // ) % numColorsInPalette;
        if (temp.real() > temp.imag())
            temp = thrust::complex<double>{ temp.real() - p.real(), p.imag() * -1 };
        else
            temp = thrust::complex<double>{ temp.imag() - p.real(), p.imag() * -1 };
        
        //temp = thrust::tanh(temp);
        thrust::complex<double> sinhTemp{ thrust::sinh(temp) };
        thrust::complex<double> coshTemp{ thrust::cosh(temp) };
        temp = sinhTemp / coshTemp;
        double tanHMagnitude{ thrust::abs(temp) };
        //if (tanHMagnitude > 5000.0) tanHMagnitude = 200.0;
        if (tanHMagnitude < 1) tanHMagnitude *= 100;
        int tanHMagnitudeNarrow{__double2int_rz(tanHMagnitude) * i};
        if (tanHMagnitudeNarrow < 0) tanHMagnitudeNarrow *= -1;
        //if (tanHMagnitude < 2147483647.0 * i) tanHMagnitudeNarrow = int(tanHMagnitude);
        //tanHMagnitudeNarrow /= 2;
        iterations[tid] = tanHMagnitudeNarrow % 5000;//tanHMagnitudeNarrow % 200; // for now, g_colorsInPalette is a constant (1024)
    }
}

// --- reserved name (for now it's just a regular julia set with no spice)
__global__ void algAirOnAJuliaString(const double* __restrict__ realCoords, const double* __restrict__ imagCoords, int* __restrict__ iterations)
{
    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Boundary check
    if (tid < g_N)
    {
        thrust::complex<double> z{ realCoords[tid], imagCoords[tid] };
        const thrust::complex<double> p{ g_juliaCenterX, g_juliaCenterY };
        int i{ 0 };
        while (i < g_maxIts)
        {
            thrust::complex<double> temp{ g_alg(z, p) };
            if (thrust::abs(temp) > g_limit * g_limit) break;   // magnitude escapes the limit?
            z = temp;
            ++i;
        }
        iterations[tid] = i;
    }
}


int TheCalcmaster(double* host_re, double* host_im, int* host_its, double limit, int maxIts, int fractalID, size_t numElements, int mode = 0, double juliaCenterX = 0.0, double juliaCenterY = 0.0, int ismove = 0)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    // Device memory allocation
    size_t doubleVectorSize = numElements * sizeof(double);
    size_t intVectorSize = numElements * sizeof(int);
    // Real coords input vector
    double* device_re = NULL;
    err = hipMalloc((void**)&device_re, doubleVectorSize);
    if (err != hipSuccess) return 4;
    // Imaginary coords input vector
    double* device_im = NULL;
    err = hipMalloc((void**)&device_im, doubleVectorSize);
    if (err != hipSuccess) return 4;
    // Iterations output vector
    int* device_its = NULL;
    err = hipMalloc((void**)&device_its, intVectorSize);
    if (err != hipSuccess) return 4;
    // Copy real coords to device
    err = hipMemcpy(device_re, host_re, doubleVectorSize, hipMemcpyHostToDevice);
    if (err != hipSuccess) return 5;
    // Copy imaginary coords to device
    err = hipMemcpy(device_im, host_im, doubleVectorSize, hipMemcpyHostToDevice);
    if (err != hipSuccess) return 5;
    // Copy the iterations to the device if we're doing a move
    if (ismove == 1)
    {
        err = hipMemcpy(device_its, host_its, intVectorSize, hipMemcpyHostToDevice);
        if (err != hipSuccess) return 5;
    }
    // Set up the thread blocks
    int threadsPerBlock{ 512 };
    int blocksPerGrid = (static_cast<int>(numElements) + threadsPerBlock - 1) / threadsPerBlock;
    // Global vars
    //setTheDeviceGlobals(double juliaCenterX, double juliaCenterY, int maxIts, double limit, int fractalFormulaID, int N)
    setTheDeviceGlobals<<<1, 1>>>(juliaCenterX, juliaCenterY, maxIts, limit, fractalID, static_cast<int>(numElements), ismove);
    //RUN THE KERNEL (const double* __restrict__ realCoords, const double* __restrict__ imagCoords, int* __restrict__ iterations)
    switch (mode)
    {
    case 1:
        algJulia<<<blocksPerGrid, threadsPerBlock>>>(device_re, device_im, device_its);
        break;
    case 2:
        algTheCalcmasterTwist<<<blocksPerGrid, threadsPerBlock>>>(device_re, device_im, device_its);
        break;
    case 3:
        algAirOnAJuliaString<<<blocksPerGrid, threadsPerBlock>>>(device_re, device_im, device_its);
        break;
    default:
        algMap<<<blocksPerGrid, threadsPerBlock>>>(device_re, device_im, device_its);
        break;
    }
    err = hipGetLastError();
    if (err != hipSuccess) return 6;
    // Copy device iterations back to m_iterations
    err = hipMemcpy(host_its, device_its, intVectorSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) return 7;
    // Step in, the janitor.
    err = hipFree(device_re);
    if (err != hipSuccess) return 8;
    err = hipFree(device_im);
    if (err != hipSuccess) return 8;
    err = hipFree(device_its);
    if (err != hipSuccess) return 8;
    return 0;
}
