#include "hip/hip_runtime.h"
// Copyright 2024 - Gregory James Carriveau a.k.a. fractalsnakes840
// MIT License
// 
// Permission is hereby granted, free of charge, to any person obtaining a copy of this
// software and associated documentation files(the �Software�), to deal in the Software
// without restriction, including without limitation the rights to use, copy, modify,
// merge, publish, distribute, sublicense, and /or sell copies of the Software, and to
// permit persons to whom the Software is furnished to do so, subject to the following conditions :
//
// The above copyright notice and this permission notice shall be included in all copies
// or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED �AS IS�, WITHOUT WARRANTY OF ANY KIND, EXPRESS OR IMPLIED,
// INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR
// PURPOSE AND NONINFRINGEMENT.IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE
// FOR ANY CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR
// OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
// DEALINGS IN THE SOFTWARE.

// Let's tickle the CUDAs :)
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <>	// not required.. I use it for getting rid of Intellisense squigglies under blockIdx, blockDim, threadIdx in Visual Studio 2022
#include <thrust/complex.h>             // numerics for double precision complex numbers

// Global device variables
__device__ const int g_colorsInPalette{ 1024 };
__device__ double g_juliaCenterX;
__device__ double g_juliaCenterY;
__device__ int    g_maxIts;
__device__ double g_limit;
__device__ int    g_N;
typedef thrust::complex<double> (*frmptr)(thrust::complex<double>, thrust::complex<double>);
__device__ frmptr g_alg;

// Fractal default 0: z^2 + p
__device__ thrust::complex<double> frmMandelbrot(thrust::complex<double> z, thrust::complex<double> p)
{
    return z * z + p;
}
// Fractal 1: (sin(z) * z)^2  + p
__device__ thrust::complex<double> frmSinPow2(thrust::complex<double> z, thrust::complex<double> p)
{
    thrust::complex<double>temp{ thrust::sin(z) * z };
    return temp * temp + p;
}
// Fractal 2: (sin(z) * z)^3  + p
__device__ thrust::complex<double> frmSinPow3(thrust::complex<double> z, thrust::complex<double> p)
{
    thrust::complex<double>temp{ thrust::sin(z) * z };
    return temp * temp * temp + p;
}
// Fractal 3: (sin(z) * z)^4  + p
__device__ thrust::complex<double> frmSinPow4(thrust::complex<double> z, thrust::complex<double> p)
{
    thrust::complex<double>temp{ thrust::sin(z) * z };
    return temp * temp * temp * temp + p;
}
// Fractal 4: (sin(z) * z / div)^2 + p
__device__ thrust::complex<double> frmSinPow2Div1(thrust::complex<double> z, thrust::complex<double> p)
{
    thrust::complex<double>temp{ z / thrust::complex<double>{4.0, -1.5} };
    temp *= thrust::sin(z);
    temp *= temp;
    return temp + p;
}
// Fractal 5: (cos(z) * z)^2  + p
__device__ thrust::complex<double> frmCosPow2(thrust::complex<double> z, thrust::complex<double> p)
{
    thrust::complex<double>temp{ thrust::cos(z) * z };
    return temp * temp + p;
}
// Fractal 6: ((z^2 + p) * (cos(z) * z)^2 + p)
__device__ thrust::complex<double> frmCosPow2MandelbrotHybrid(thrust::complex<double> z, thrust::complex<double> p)
{
    thrust::complex<double>temp{ z * z + p};
    thrust::complex<double>temp2{ thrust::cos(z) * z };
    temp2 = temp2 * temp2 + p;
    return temp * temp2;
}
// Fractal 7: Mandelbrot => Burning Ship (+re, -im) => (cos(Burning Ship) * (Burning Ship))^2 + p
__device__ thrust::complex<double> frmCosPow2AbsRIMandelbrotHybrid(thrust::complex<double> z, thrust::complex<double> p)
{
    thrust::complex<double>temp{ thrust::cos(z) * z };
    // this is the conjugate of Mandelbrot z
    temp = thrust::complex<double>{ temp.real(), -temp.imag() };
    // Hybridization of Burning Ship
    temp = thrust::cos(temp) * temp; // cos(+re,-im) * (+re,-im)
    return temp * temp + p;
}
// Fractal 8: (cos(+re,-im) * (+re,-im))^2 + p
__device__ thrust::complex<double> frmCosPow2AbsRI(thrust::complex<double> z, thrust::complex<double> p)
{
    thrust::complex<double> temp{ abs(z.real()), -abs(z.imag()) };
    // this is the conjugate of Mandelbrot z
    temp = thrust::cos(temp) * temp;
    return temp * temp + p;
}
// Fractal 9: Experiment number 1 ((1 - z^3) / 6) / ((1 - z - z^2) / 2)^2 + p
__device__ thrust::complex<double> frmExperiment1(thrust::complex<double> z, thrust::complex<double> p)
{
    thrust::complex<double> temp{ (1 - z * z * z) / 6 };
    thrust::complex<double> temp2{ (1 - z - z * z) / 2 };
    temp /= temp2 * temp2;
    return temp + p;
}

__global__ void setTheDeviceGlobals(double juliaCenterX, double juliaCenterY, int maxIts, double limit, int fractalFormulaID, int N)
{
    g_juliaCenterX = juliaCenterX;
    g_juliaCenterY = juliaCenterY;
    g_maxIts = maxIts;
    g_limit = limit;
    g_N = N;
    switch (fractalFormulaID)
    {
    case 1:
        g_alg = frmSinPow2;
        break;
    case 2:
        g_alg = frmSinPow3;
        break;
    case 3:
        g_alg = frmSinPow4;
        break;
    case 4:
        g_alg = frmSinPow2Div1;
        break;
    case 5:
        g_alg = frmCosPow2;
        break;
    case 6:
        g_alg = frmCosPow2MandelbrotHybrid;
        break;
    case 7:
        g_alg = frmCosPow2AbsRIMandelbrotHybrid;
        break;
    case 8:
        g_alg = frmCosPow2AbsRI;
        break;
    case 9:
        g_alg = frmExperiment1;
        break;
    default:
        g_alg = frmMandelbrot;
        break;
    }
}

__global__ void algMap(const double* __restrict__ realCoords, const double* __restrict__ imagCoords, int* __restrict__ iterations)
{
    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Boundary check
    if (tid < g_N)
    {
        thrust::complex<double> z{ 0.0,0.0 };
        const thrust::complex<double> p{ realCoords[tid], imagCoords[tid] };
        int i{ 0 };
        while (i < g_maxIts)
        {
            thrust::complex<double> temp{ g_alg(z, p)};
            if (thrust::abs(temp) > g_limit * g_limit) break;   // magnitude escapes the limit?
            z = temp;
            ++i;
        }
        iterations[tid] = i;
    }
}

__global__ void algJulia(const double* __restrict__ realCoords, const double* __restrict__ imagCoords, int* __restrict__ iterations)
{
    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Boundary check
    if (tid < g_N)
    {
        thrust::complex<double> z{ realCoords[tid], imagCoords[tid] };
        const thrust::complex<double> p{ g_juliaCenterX, g_juliaCenterY };
        int i{ 0 };
        while (i < g_maxIts)
        {
            thrust::complex<double> temp{ g_alg(z, p) };
            if (thrust::abs(temp) > g_limit * g_limit) break;   // magnitude escapes the limit?
            z = temp;
            ++i;
        }
        iterations[tid] = i;
    }
}

// Pretty leaves and scary hairy eyeball videos in areas near black holes
__global__ void algTheCalcmasterTwist(const double* __restrict__ realCoords, const double* __restrict__ imagCoords, int* __restrict__ iterations)
{
    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Boundary check
    if (tid < g_N)
    {
        thrust::complex<double> z{ realCoords[tid], imagCoords[tid] };
        const thrust::complex<double> p{ g_juliaCenterX, g_juliaCenterY };
        int i{ 0 };
        while (i < g_maxIts)
        {
            thrust::complex<double> temp{ g_alg(z, p) };
            if (thrust::abs(temp) > g_limit * g_limit) break;   // magnitude escapes the limit?
            z = temp;
            ++i;
        }
        //return (Int32)(Complex.Tanh(Complex.Subtract(Math.Min(res.Real, res.Imaginary), p)).Magnitude * its) % numColorsInPalette;
        thrust::complex<double> minReImLessP{ z.real() > z.imag() ? z.imag() - p : z.real() - p };
        double tanHMagnitude{ tanh(thrust::abs(minReImLessP)) };
        iterations[tid] = static_cast<int>(tanHMagnitude) * i % g_colorsInPalette; // for now, g_colorsInPalette is a constant (1024)
    }
}

// --- reserved name (for now it's just a regular julia set with no spice)
__global__ void algAirOnAJuliaString(const double* __restrict__ realCoords, const double* __restrict__ imagCoords, int* __restrict__ iterations)
{
    // Calculate global thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Boundary check
    if (tid < g_N)
    {
        thrust::complex<double> z{ realCoords[tid], imagCoords[tid] };
        const thrust::complex<double> p{ g_juliaCenterX, g_juliaCenterY };
        int i{ 0 };
        while (i < g_maxIts)
        {
            thrust::complex<double> temp{ g_alg(z, p) };
            if (thrust::abs(temp) > g_limit * g_limit) break;   // magnitude escapes the limit?
            z = temp;
            ++i;
        }
        iterations[tid] = i;
    }
}


int TheCalcmaster(double* host_re, double* host_im, int* host_its, double limit, int maxIts, int fractalID, size_t numElements, int mode = 0, double juliaCenterX = 0.0, double juliaCenterY = 0.0)
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    // Device memory allocation
    size_t doubleVectorSize = numElements * sizeof(double);
    size_t intVectorSize = numElements * sizeof(int);
    // Real coords input vector
    double* device_re = NULL;
    err = hipMalloc((void**)&device_re, doubleVectorSize);
    if (err != hipSuccess) return 4;
    // Imaginary coords input vector
    double* device_im = NULL;
    err = hipMalloc((void**)&device_im, doubleVectorSize);
    if (err != hipSuccess) return 4;
    // Iterations output vector
    int* device_its = NULL;
    err = hipMalloc((void**)&device_its, intVectorSize);
    if (err != hipSuccess) return 4;
    // Copy real coords to device
    err = hipMemcpy(device_re, host_re, doubleVectorSize, hipMemcpyHostToDevice);
    if (err != hipSuccess) return 5;
    // Copy imaginary coords to device
    err = hipMemcpy(device_im, host_im, doubleVectorSize, hipMemcpyHostToDevice);
    if (err != hipSuccess) return 5;
    // Set up the thread blocks
    int threadsPerBlock{ 512 };
    int blocksPerGrid = (static_cast<int>(numElements) + threadsPerBlock - 1) / threadsPerBlock;
    // Global vars
    //setTheDeviceGlobals(double juliaCenterX, double juliaCenterY, int maxIts, double limit, int fractalFormulaID, int N)
    setTheDeviceGlobals<<<1, 1>>>(juliaCenterX, juliaCenterY, maxIts, limit, fractalID, static_cast<int>(numElements));
    //RUN THE KERNEL (const double* __restrict__ realCoords, const double* __restrict__ imagCoords, int* __restrict__ iterations)
    switch (mode)
    {
    case 1:
        algJulia<<<blocksPerGrid, threadsPerBlock>>>(device_re, device_im, device_its);
        break;
    case 2:
        algTheCalcmasterTwist<<<blocksPerGrid, threadsPerBlock>>>(device_re, device_im, device_its);
        break;
    case 3:
        algAirOnAJuliaString<<<blocksPerGrid, threadsPerBlock>>>(device_re, device_im, device_its);
        break;
    default:
        algMap<<<blocksPerGrid, threadsPerBlock>>>(device_re, device_im, device_its);
        break;
    }
    err = hipGetLastError();
    if (err != hipSuccess) return 6;
    // Copy device iterations back to m_iterations
    err = hipMemcpy(host_its, device_its, intVectorSize, hipMemcpyDeviceToHost);
    if (err != hipSuccess) return 7;
    // Step in, the janitor.
    err = hipFree(device_re);
    if (err != hipSuccess) return 8;
    err = hipFree(device_im);
    if (err != hipSuccess) return 8;
    err = hipFree(device_its);
    if (err != hipSuccess) return 8;
    return 0;
}
